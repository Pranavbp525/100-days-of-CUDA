#include "hip/hip_runtime.h"
#include <iostream>
#include <climits> 
#include <hip/hip_runtime.h>
using namespace std;

struct CSRGraph {
    unsigned int numVertices;
    unsigned int* srcPtrs;  
    unsigned int* dst;      
};

__global__ void bfs_kernel(CSRGraph csrGraph, unsigned int* level, unsigned int *prevFrontier, unsigned int *currFrontier, unsigned int* numprevFrontier, unsigned int* numcurrFrontier, unsigned int currLevel) {
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < numprevFrontier) {
        unsigned int vertex = prevFrontier[i];
         
        for (unsigned int edge = csrGraph.srcPtrs[vertex]; edge < csrGraph.srcPtrs[vertex + 1]; ++edge) {
            unsigned int neighbor = csrGraph.dst[edge];
            if (atomicCAS(&level[neighbor], UINT_MAX, currLevel)==UINT_MAX) {
                unsigned int currFrontierIdx = atomicAdd(numcurrFrontier, 1);
                currFrontier[currFrontierIdx] = neighbor;
            }
        }
        
    }
}

void bfs_host(CSRGraph csrGraph_h, unsigned int startVertex, unsigned int* level_h) {
    unsigned int *d_srcPtrs, *d_dst, *d_level, *d_prevFrontier, *d_currFrontier, *d_numprevFrontier, *d_numcurrFrontier;
    unsigned int numVertices = csrGraph_h.numVertices;

    hipMalloc(&d_srcPtrs, (numVertices + 1) * sizeof(unsigned int));
    hipMalloc(&d_dst, csrGraph_h.srcPtrs[numVertices] * sizeof(unsigned int));  // Total edges
    hipMalloc(&d_level, numVertices * sizeof(unsigned int));

    hipMalloc(&d_prevFrontier, numVertices * sizeof(unsigned int));
    hipMalloc(&d_currFrontier, numVertices * sizeof(unsigned int));
    hipMalloc(&d_numprevFrontier, sizeof(unsigned int));
    hipMalloc(&d_numcurrFrontier, sizeof(unsigned int));

    hipMemcpy(d_srcPtrs, csrGraph_h.srcPtrs, (numVertices + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_dst, csrGraph_h.dst, csrGraph_h.srcPtrs[numVertices] * sizeof(unsigned int), hipMemcpyHostToDevice);

    for (unsigned int i = 0; i < numVertices; i++) {
        level_h[i] = (i == startVertex) ? 0 : UINT_MAX;
    }
    hipMemcpy(d_level, level_h, numVertices * sizeof(unsigned int), hipMemcpyHostToDevice);

    CSRGraph csrGraph_d;
    csrGraph_d.numVertices = numVertices;
    csrGraph_d.srcPtrs = d_srcPtrs;
    csrGraph_d.dst = d_dst;

    CSRGraph *d_csrGraph;
    hipMalloc(&d_csrGraph, sizeof(CSRGraph));
    hipMemcpy(d_csrGraph, &csrGraph_d, sizeof(CSRGraph), hipMemcpyHostToDevice);

    unsigned int h_numprevFrontier = 1;
    unsigned int h_numcurrFrontier = 0;
    unsigned int h_prevFrontier[1] = {startVertex};
    hipMemcpy(d_prevFrontier, h_prevFrontier, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_numprevFrontier, &h_numprevFrontier, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_numcurrFrontier, &h_numcurrFrontier, sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int currLevel = 1;
    while (h_numprevFrontier > 0) {
        int blockSize = 256;
        int numBlocks = (h_numprevFrontier + blockSize - 1) / blockSize;

        bfs_kernel<<<numBlocks, blockSize>>>(*d_csrGraph, d_level, d_prevFrontier, d_currFrontier, d_numprevFrontier, d_numcurrFrontier, currLevel);
        hipDeviceSynchronize();  

        unsigned int* temp = d_prevFrontier;
        d_prevFrontier = d_currFrontier;
        d_currFrontier = temp;

        hipMemcpy(&h_numprevFrontier, d_numcurrFrontier, sizeof(unsigned int), hipMemcpyDeviceToHost);
        h_numcurrFrontier = 0;
        hipMemcpy(d_numcurrFrontier, &h_numcurrFrontier, sizeof(unsigned int), hipMemcpyHostToDevice);

        currLevel++;
    }

    hipMemcpy(level_h, d_level, numVertices * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(d_srcPtrs);
    hipFree(d_dst);
    hipFree(d_level);
    hipFree(d_prevFrontier);
    hipFree(d_currFrontier);
    hipFree(d_numprevFrontier);
    hipFree(d_numcurrFrontier);
    hipFree(d_csrGraph);
}

int main() {
    unsigned int numVertices = 4;
    unsigned int srcPtrs_h[] = {0, 1, 2, 3, 4}; 
    unsigned int dst_h[] = {1, 2, 3, 0};         

    CSRGraph csrGraph_h;
    csrGraph_h.numVertices = numVertices;
    csrGraph_h.srcPtrs = srcPtrs_h;
    csrGraph_h.dst = dst_h;

    unsigned int startVertex = 0;
    unsigned int level_h[4];

    bfs_host(csrGraph_h, startVertex, level_h);

    for (unsigned int i = 0; i < numVertices; i++) {
        cout << "Vertex " << i << ": Level " << level_h[i] << endl;
    }

    return 0;
}