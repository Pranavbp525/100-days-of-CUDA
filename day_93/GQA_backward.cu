#include <iostream>
#include <vector>
#include <cmath>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call) { \
    const hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "cublasStatus: %d\n", status); \
        exit(1); \
    } \
}


__global__ void find_max_kernel(const float* input, float* max_vals, int N, int D) {
    int n = blockIdx.x;
    extern __shared__ float sdata[];
    sdata[threadIdx.x] = -INFINITY;
    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        sdata[threadIdx.x] = fmaxf(sdata[threadIdx.x], input[n * D + i]);
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) sdata[threadIdx.x] = fmaxf(sdata[threadIdx.x], sdata[threadIdx.x + s]);
        __syncthreads();
    }
    if (threadIdx.x == 0) max_vals[n] = sdata[0];
}

__global__ void exp_sum_normalize_kernel(float* data, const float* max_vals, float* sum_vals, int N, int D) {
    int n = blockIdx.x;
    extern __shared__ float sdata[];
    float max_val = max_vals[n];
    sdata[threadIdx.x] = 0.0f;
    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        float val = expf(data[n * D + i] - max_val);
        data[n * D + i] = val;
        sdata[threadIdx.x] += val;
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }
    if (threadIdx.x == 0) sum_vals[n] = sdata[0];
    __syncthreads();
    float sum_val = sum_vals[n];
    if (sum_val > 0) {
        for (int i = threadIdx.x; i < D; i += blockDim.x) {
            data[n * D + i] /= sum_val;
        }
    }
}

__global__ void scale_kernel(float* matrix, float scale_factor, int n_elements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_elements) {
        matrix[i] *= scale_factor;
    }
}

__global__ void concat_heads_kernel(const float* input, float* output, int N, int H, int L, int Dh) {
    int n = blockIdx.x / L;
    int l = blockIdx.x % L;
    
    for (int h = threadIdx.y; h < H; h += blockDim.y) {
        for (int d = threadIdx.x; d < Dh; d += blockDim.x) {
            int input_idx = n * H * L * Dh + h * L * Dh + l * Dh + d;
            int output_idx = n * L * (H * Dh) + l * (H * Dh) + h * Dh + d;
            output[output_idx] = input[input_idx];
        }
    }
}

__global__ void split_heads_grad_kernel(const float* grad_in, float* grad_out, int N, int H, int L, int Dh) {
    int n = blockIdx.x / L;
    int l = blockIdx.x % L;
    
    for (int h = threadIdx.y; h < H; h += blockDim.y) {
        for (int d = threadIdx.x; d < Dh; d += blockDim.x) {
            int input_idx = n * L * (H * Dh) + l * (H * Dh) + h * Dh + d;
            int output_idx = n * H * L * Dh + h * L * Dh + l * Dh + d;
            grad_out[output_idx] = grad_in[input_idx];
        }
    }
}

__global__ void softmax_backward_kernel(float* d_scores, const float* softmax_output, const float* d_softmax_scores, int N, int D) {
    int n = blockIdx.x;
    extern __shared__ float sdata[];

    float dot_product = 0.0f;
    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        dot_product += d_softmax_scores[n * D + i] * softmax_output[n * D + i];
    }
    sdata[threadIdx.x] = dot_product;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }
    dot_product = sdata[0];
    __syncthreads();

    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        int idx = n * D + i;
        d_scores[idx] = softmax_output[idx] * (d_softmax_scores[idx] - dot_product);
    }
}

__global__ void add_matrices_kernel(float* out, const float* in1, const float* in2, const float* in3, int n_elements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_elements) {
        out[i] = in1[i] + in2[i] + in3[i];
    }
}


void print_matrix(const std::string& name, const float* m, int rows, int cols) {
    std::cout << "--- " << name << " --- (" << rows << "x" << cols << ")\n";
    std::vector<float> h_m(rows * cols);
    CHECK_CUDA(hipMemcpy(h_m.data(), m, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < std::min(4, rows); ++i) {
        for (int j = 0; j < std::min(8, cols); ++j) {
            std::cout << h_m[i * cols + j] << "\t";
        }
        std::cout << (cols > 8 ? "...\n" : "\n");
    }
    std::cout << (rows > 4 ? "...\n" : "");
    std::cout << "-----------------------\n" << std::endl;
}


int main() {
    const int batch_size = 32;       
    const int seq_len = 64;          
    const int embed_dim = 512;       
    const int num_q_heads = 8;
    const int num_kv_heads = 2;
    
    if (embed_dim % num_q_heads != 0 || num_q_heads % num_kv_heads != 0) {
        std::cerr << "Dimension or head configuration is invalid." << std::endl;
        return 1;
    }
    const int head_dim = embed_dim / num_q_heads;
    const int heads_per_group = num_q_heads / num_kv_heads;

    
    std::vector<float> h_input(batch_size * seq_len * embed_dim);
    std::vector<float> h_Wq(embed_dim * embed_dim);
    std::vector<float> h_Wk(embed_dim * (num_kv_heads * head_dim));
    std::vector<float> h_Wv(embed_dim * (num_kv_heads * head_dim));
    std::vector<float> h_Wo(embed_dim * embed_dim);
    std::vector<float> h_grad_output(h_input.size());

    std::mt19937 gen(42);
    std::normal_distribution<float> dis(0.0f, 0.02f);
    for (float& v : h_input) v = dis(gen);
    for (float& v : h_Wq) v = dis(gen); for (float& v : h_Wk) v = dis(gen);
    for (float& v : h_Wv) v = dis(gen); for (float& v : h_Wo) v = dis(gen);
    for (float& v : h_grad_output) v = dis(gen);

    float *d_input, *d_Wq, *d_Wk, *d_Wv, *d_Wo, *d_Q, *d_K_gqa, *d_V_gqa;
    float *d_scores, *d_softmax_scores, *d_attn_heads_output, *d_concat_heads, *d_output;
    float *d_softmax_max, *d_softmax_sum;
    float *d_grad_output, *d_grad_concat_heads, *d_grad_Wo, *d_grad_attn_heads_output;
    float *d_grad_V_gqa, *d_grad_softmax_scores, *d_grad_scores, *d_grad_Q, *d_grad_K_gqa;
    float *d_grad_Wq, *d_grad_Wk, *d_grad_Wv, *d_grad_input;
    float *d_grad_input_from_Q, *d_grad_input_from_K, *d_grad_input_from_V;

    CHECK_CUDA(hipMalloc(&d_input, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wq, h_Wq.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wk, h_Wk.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wv, h_Wv.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wo, h_Wo.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Q, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_K_gqa, batch_size * seq_len * num_kv_heads * head_dim * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_V_gqa, batch_size * seq_len * num_kv_heads * head_dim * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_scores, batch_size * num_q_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_softmax_scores, batch_size * num_q_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_heads_output, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_concat_heads, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_softmax_max, batch_size * num_q_heads * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_softmax_sum, batch_size * num_q_heads * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_output, h_grad_output.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_concat_heads, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wo, h_Wo.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_attn_heads_output, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_V_gqa, batch_size * seq_len * num_kv_heads * head_dim * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_softmax_scores, batch_size * num_q_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_scores, batch_size * num_q_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Q, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_K_gqa, batch_size * seq_len * num_kv_heads * head_dim * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wq, h_Wq.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wk, h_Wk.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wv, h_Wv.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input_from_Q, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input_from_K, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input_from_V, h_input.size() * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wq, h_Wq.data(), h_Wq.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wk, h_Wk.data(), h_Wk.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wv, h_Wv.data(), h_Wv.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wo, h_Wo.data(), h_Wo.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_grad_output, h_grad_output.data(), h_grad_output.size() * sizeof(float), hipMemcpyHostToDevice));
    
    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));

    const float alpha = 1.0f, beta = 0.0f, beta_one = 1.0f;
    const int N_in = batch_size * seq_len;
    const int kv_embed_dim = num_kv_heads * head_dim;

    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wq, embed_dim, d_input, embed_dim, &beta, d_Q, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, kv_embed_dim, N_in, embed_dim, &alpha, d_Wk, embed_dim, d_input, embed_dim, &beta, d_K_gqa, kv_embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, kv_embed_dim, N_in, embed_dim, &alpha, d_Wv, embed_dim, d_input, embed_dim, &beta, d_V_gqa, kv_embed_dim));
    
    long long int kv_head_stride = seq_len * head_dim;
    long long int q_head_stride = seq_len * head_dim;
    for (int i = 0; i < num_kv_heads; ++i) {
        CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, seq_len, seq_len, head_dim, &alpha, d_K_gqa + i * kv_head_stride, head_dim, 0, d_Q + i * heads_per_group * q_head_stride, head_dim, q_head_stride, &beta, d_scores + i * heads_per_group * seq_len * seq_len, seq_len, seq_len * seq_len, batch_size * heads_per_group));
    }
    
    const float scale_factor = 1.0f / sqrtf((float)head_dim);
    scale_kernel<<<(batch_size * num_q_heads * seq_len * seq_len + 255) / 256, 256>>>(d_scores, scale_factor, batch_size * num_q_heads * seq_len * seq_len);
    
    CHECK_CUDA(hipMemcpy(d_softmax_scores, d_scores, batch_size * num_q_heads * seq_len * seq_len * sizeof(float), hipMemcpyDeviceToDevice));
    int softmax_rows = batch_size * num_q_heads * seq_len;
    find_max_kernel<<<softmax_rows, 256, 256 * sizeof(float)>>>(d_softmax_scores, d_softmax_max, softmax_rows, seq_len);
    exp_sum_normalize_kernel<<<softmax_rows, 256, 256 * sizeof(float)>>>(d_softmax_scores, d_softmax_max, d_softmax_sum, softmax_rows, seq_len);
    
    for (int i = 0; i < num_kv_heads; ++i) {
        CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, head_dim, seq_len, seq_len, &alpha, d_V_gqa + i * kv_head_stride, head_dim, 0, d_softmax_scores + i * heads_per_group * seq_len * seq_len, seq_len, seq_len * seq_len, &beta, d_attn_heads_output + i * heads_per_group * q_head_stride, head_dim, q_head_stride, batch_size * heads_per_group));
    }
    
    dim3 concat_blocks(N_in), concat_threads(32, 8);
    concat_heads_kernel<<<concat_blocks, concat_threads>>>(d_attn_heads_output, d_concat_heads, batch_size, num_q_heads, seq_len, head_dim);
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wo, embed_dim, d_concat_heads, embed_dim, &beta, d_output, embed_dim));

    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, embed_dim, N_in, &alpha, d_grad_output, embed_dim, d_concat_heads, embed_dim, &beta, d_grad_Wo, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, embed_dim, N_in, embed_dim, &alpha, d_Wo, embed_dim, d_grad_output, embed_dim, &beta, d_grad_concat_heads, embed_dim));
    split_heads_grad_kernel<<<concat_blocks, concat_threads>>>(d_grad_concat_heads, d_grad_attn_heads_output, batch_size, num_q_heads, seq_len, head_dim);

    CHECK_CUDA(hipMemset(d_grad_V_gqa, 0, batch_size * seq_len * num_kv_heads * head_dim * sizeof(float)));
    CHECK_CUDA(hipMemset(d_grad_K_gqa, 0, batch_size * seq_len * num_kv_heads * head_dim * sizeof(float)));

    for (int i = 0; i < num_kv_heads; ++i) {
        CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, seq_len, seq_len, head_dim, &alpha, d_grad_attn_heads_output + i * heads_per_group * q_head_stride, head_dim, q_head_stride, d_V_gqa + i * kv_head_stride, head_dim, 0, &beta, d_grad_softmax_scores + i * heads_per_group * seq_len * seq_len, seq_len, seq_len*seq_len, batch_size * heads_per_group));
        CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, head_dim, seq_len, seq_len, &alpha, d_grad_attn_heads_output + i * heads_per_group * q_head_stride, head_dim, q_head_stride, d_softmax_scores + i * heads_per_group * seq_len * seq_len, seq_len, seq_len*seq_len, &beta_one, d_grad_V_gqa + i * kv_head_stride, head_dim, kv_head_stride, batch_size * heads_per_group));
    }
    
    softmax_backward_kernel<<<softmax_rows, 256, 256*sizeof(float)>>>(d_grad_scores, d_softmax_scores, d_grad_softmax_scores, softmax_rows, seq_len);
    scale_kernel<<<(batch_size * num_q_heads * seq_len * seq_len + 255) / 256, 256>>>(d_grad_scores, scale_factor, batch_size * num_q_heads * seq_len * seq_len);
    
    for (int i = 0; i < num_kv_heads; ++i) {
        CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, head_dim, seq_len, seq_len, &alpha, d_K_gqa + i * kv_head_stride, head_dim, 0, d_grad_scores + i * heads_per_group * seq_len * seq_len, seq_len, seq_len*seq_len, &beta, d_grad_Q + i * heads_per_group * q_head_stride, head_dim, q_head_stride, batch_size * heads_per_group));
        CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, head_dim, seq_len, seq_len, &alpha, d_Q + i * heads_per_group * q_head_stride, head_dim, q_head_stride, d_grad_scores + i * heads_per_group * seq_len * seq_len, seq_len, seq_len*seq_len, &beta_one, d_grad_K_gqa + i * kv_head_stride, head_dim, kv_head_stride, batch_size * heads_per_group));
    }
    
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embed_dim, embed_dim, N_in, &alpha, d_grad_Q, embed_dim, d_input, embed_dim, &beta, d_grad_Wq, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, kv_embed_dim, embed_dim, N_in, &alpha, d_grad_K_gqa, kv_embed_dim, d_input, embed_dim, &beta, d_grad_Wk, kv_embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, kv_embed_dim, embed_dim, N_in, &alpha, d_grad_V_gqa, kv_embed_dim, d_input, embed_dim, &beta, d_grad_Wv, kv_embed_dim));
    
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wq, embed_dim, d_grad_Q, embed_dim, &beta, d_grad_input_from_Q, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, N_in, kv_embed_dim, &alpha, d_Wk, embed_dim, d_grad_K_gqa, kv_embed_dim, &beta, d_grad_input_from_K, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, N_in, kv_embed_dim, &alpha, d_Wv, embed_dim, d_grad_V_gqa, kv_embed_dim, &beta, d_grad_input_from_V, embed_dim));
    add_matrices_kernel<<<(h_input.size() + 255) / 256, 256>>>(d_grad_input, d_grad_input_from_Q, d_grad_input_from_K, d_grad_input_from_V, h_input.size());

    std::cout << "Backward pass complete." << std::endl;
    print_matrix("Gradient w.r.t. Wq", d_grad_Wq, embed_dim, embed_dim);
    print_matrix("Gradient w.r.t. Wk", d_grad_Wk, embed_dim, kv_embed_dim);
    print_matrix("Gradient w.r.t. Input", d_grad_input, N_in, embed_dim);


    CHECK_CUBLAS(hipblasDestroy(cublas_handle));

    return 0;
}
