#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

struct COOMatrix {
    unsigned int* rowIdx;      
    unsigned int* colIdx;      
    float* values;             
    unsigned int numNonZeros;  
    unsigned int numRows;      
    unsigned int numCols;      
};

__global__ void spmv_coo_kernel(COOMatrix cooMatrix, float *x, float *y) {
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < cooMatrix.numNonZeros) {
        unsigned int row = cooMatrix.rowIdx[i];
        unsigned int col = cooMatrix.colIdx[i];  
        float value = cooMatrix.values[i];       
        atomicAdd(&y[row], x[col] * value);
    }
}

void spmv_coo_host(COOMatrix cooMatrix_h, float* x_h, float* y_h) {
    unsigned int *d_rowIdx, *d_colIdx;
    float *d_values, *d_x, *d_y;

    hipMalloc(&d_rowIdx, cooMatrix_h.numNonZeros * sizeof(unsigned int));
    hipMalloc(&d_colIdx, cooMatrix_h.numNonZeros * sizeof(unsigned int));
    hipMalloc(&d_values, cooMatrix_h.numNonZeros * sizeof(float));
    hipMalloc(&d_x, cooMatrix_h.numCols * sizeof(float));
    hipMalloc(&d_y, cooMatrix_h.numRows * sizeof(float));

    hipMemcpy(d_rowIdx, cooMatrix_h.rowIdx, cooMatrix_h.numNonZeros * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, cooMatrix_h.colIdx, cooMatrix_h.numNonZeros * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, cooMatrix_h.values, cooMatrix_h.numNonZeros * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x_h, cooMatrix_h.numCols * sizeof(float), hipMemcpyHostToDevice);

    hipMemset(d_y, 0, cooMatrix_h.numRows * sizeof(float));

    COOMatrix cooMatrix_d;
    cooMatrix_d.rowIdx = d_rowIdx;
    cooMatrix_d.colIdx = d_colIdx;
    cooMatrix_d.values = d_values;
    cooMatrix_d.numNonZeros = cooMatrix_h.numNonZeros;
    cooMatrix_d.numRows = cooMatrix_h.numRows;
    cooMatrix_d.numCols = cooMatrix_h.numCols;

    COOMatrix *d_cooMatrix;
    hipMalloc(&d_cooMatrix, sizeof(COOMatrix));
    hipMemcpy(d_cooMatrix, &cooMatrix_d, sizeof(COOMatrix), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (cooMatrix_h.numNonZeros + blockSize - 1) / blockSize;

    spmv_coo_kernel<<<numBlocks, blockSize>>>(*d_cooMatrix, d_x, d_y);

    hipDeviceSynchronize();

    hipMemcpy(y_h, d_y, cooMatrix_h.numRows * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_rowIdx);
    hipFree(d_colIdx);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_cooMatrix);
}

int main() {
    // Example: 2x2 matrix in COO format
    // Matrix: [1 2]
    //         [0 3]
    unsigned int rowIdx_h[] = {0, 0, 1};  
    unsigned int colIdx_h[] = {0, 1, 1};  
    float values_h[] = {1.0, 2.0, 3.0};   
    unsigned int numNonZeros = 3;
    unsigned int numRows = 2;
    unsigned int numCols = 2;

    float x_h[] = {1.0, 2.0};  

    float y_h[2] = {0.0, 0.0};  

    COOMatrix cooMatrix_h;
    cooMatrix_h.rowIdx = rowIdx_h;
    cooMatrix_h.colIdx = colIdx_h;
    cooMatrix_h.values = values_h;
    cooMatrix_h.numNonZeros = numNonZeros;
    cooMatrix_h.numRows = numRows;
    cooMatrix_h.numCols = numCols;

    spmv_coo_host(cooMatrix_h, x_h, y_h);

    cout << "Result vector y:" << endl;
    for (int i = 0; i < numRows; i++) {
        cout << y_h[i] << " ";
    }
    cout << endl;

    return 0;
}