#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

__global__ void vectorAddKernel(float* A, float* B, float* C, int n){

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i<n){

        C[i] = A[i] + B[i];

    }

}

void vector_add(float *A_h, float *B_h, float *C_h, int n) {

    int size = n * sizeof(float);

    float *A_d, *B_d, *C_d;

    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    int block_size = 256.0;
    int grid_size = (n + block_size - 1) / block_size;

    vectorAddKernel<<grid_size, block_size>>(A_d, B_d, C_d, n);

    hipDeviceSynchronize();

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d); 


}

int main(){

    int n = 10000;
    int size = n * sizeof(float);

    float* A_h = new float[n];
    float* B_h = new float[n];
    float* C_h = new float[n]; 
    
    


}