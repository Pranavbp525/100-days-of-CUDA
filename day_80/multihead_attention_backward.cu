#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call) { \
    const hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "cublasStatus: %d\n", status); \
        exit(1); \
    } \
}

// softmax & scaling kernels

__global__ void find_max_kernel(const float* input, float* max_vals, int N, int D) {
    int n = blockIdx.x;
    extern __shared__ float sdata[];
    sdata[threadIdx.x] = -INFINITY;
    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        sdata[threadIdx.x] = fmaxf(sdata[threadIdx.x], input[n * D + i]);
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) sdata[threadIdx.x] = fmaxf(sdata[threadIdx.x], sdata[threadIdx.x + s]);
        __syncthreads();
    }
    if (threadIdx.x == 0) max_vals[n] = sdata[0];
}

__global__ void exp_sum_normalize_kernel(float* data, const float* max_vals, float* sum_vals, int N, int D) {
    int n = blockIdx.x;
    extern __shared__ float sdata[];
    float max_val = max_vals[n];
    sdata[threadIdx.x] = 0.0f;
    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        float val = expf(data[n * D + i] - max_val);
        data[n * D + i] = val;
        sdata[threadIdx.x] += val;
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }
    if (threadIdx.x == 0) sum_vals[n] = sdata[0];
    __syncthreads();
    float sum_val = sum_vals[n];
    if (sum_val > 0) {
        for (int i = threadIdx.x; i < D; i += blockDim.x) {
            data[n * D + i] /= sum_val;
        }
    }
}

__global__ void scale_kernel(float* matrix, float scale_factor, int n_elements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_elements) {
        matrix[i] *= scale_factor;
    }
}

__global__ void concat_heads_kernel(const float* input, float* output, int N, int H, int L, int Dh) {
    int n = blockIdx.x / L;
    int l = blockIdx.x % L;
    
    for (int h = threadIdx.y; h < H; h += blockDim.y) {
        for (int d = threadIdx.x; d < Dh; d += blockDim.x) {
            int input_idx = n * H * L * Dh + h * L * Dh + l * Dh + d;
            int output_idx = n * L * (H * Dh) + l * (H * Dh) + h * Dh + d;
            output[output_idx] = input[input_idx];
        }
    }
}

// backward pass kernels

// Inverse of concat_heads_kernel
__global__ void split_heads_grad_kernel(const float* grad_in, float* grad_out, int N, int H, int L, int Dh) {
    int n = blockIdx.x / L;
    int l = blockIdx.x % L;
    
    for (int h = threadIdx.y; h < H; h += blockDim.y) {
        for (int d = threadIdx.x; d < Dh; d += blockDim.x) {
            int input_idx = n * L * (H * Dh) + l * (H * Dh) + h * Dh + d;
            int output_idx = n * H * L * Dh + h * L * Dh + l * Dh + d;
            grad_out[output_idx] = grad_in[input_idx];
        }
    }
}

__global__ void softmax_backward_kernel(float* d_scores, const float* softmax_output, const float* d_softmax_scores, int N, int D) {
    int n = blockIdx.x;
    extern __shared__ float sdata[];

    float dot_product = 0.0f;
    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        dot_product += d_softmax_scores[n * D + i] * softmax_output[n * D + i];
    }
    sdata[threadIdx.x] = dot_product;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }
    dot_product = sdata[0];
    __syncthreads();

    for (int i = threadIdx.x; i < D; i += blockDim.x) {
        int idx = n * D + i;
        d_scores[idx] = softmax_output[idx] * (d_softmax_scores[idx] - dot_product);
    }
}

__global__ void add_matrices_kernel(float* out, const float* in1, const float* in2, const float* in3, int n_elements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_elements) {
        out[i] = in1[i] + in2[i] + in3[i];
    }
}

void print_matrix(const std::string& name, const float* m, int rows, int cols) {
    std::cout << "--- " << name << " --- (" << rows << "x" << cols << ")\n";
    std::vector<float> h_m(rows * cols);
    CHECK_CUDA(hipMemcpy(h_m.data(), m, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < std::min(4, rows); ++i) {
        for (int j = 0; j < std::min(8, cols); ++j) {
            std::cout << h_m[i * cols + j] << "\t";
        }
        std::cout << (cols > 8 ? "...\n" : "\n");
    }
    std::cout << (rows > 4 ? "...\n" : "");
    std::cout << "-----------------------\n" << std::endl;
}


int main() {
    // Multi-Head Attention Dimensions
    const int batch_size = 32;
    const int seq_len = 64;
    const int embed_dim = 512;
    const int num_heads = 8;
    
    if (embed_dim % num_heads != 0) {
        std::cerr << "Embedding dimension must be divisible by the number of heads." << std::endl;
        return 1;
    }
    const int head_dim = embed_dim / num_heads;

    std::cout << "--- Multi-Head Attention Forward & Backward Pass ---" << std::endl;

    std::vector<float> h_input(batch_size * seq_len * embed_dim);
    std::vector<float> h_Wq(embed_dim * embed_dim), h_Wk(embed_dim * embed_dim), h_Wv(embed_dim * embed_dim), h_Wo(embed_dim * embed_dim);
    std::vector<float> h_grad_output(h_input.size());

    std::mt19937 gen(42);
    std::normal_distribution<float> dis(0.0f, 0.02f);
    for (float& v : h_input) v = dis(gen);
    for (float& v : h_Wq) v = dis(gen);
    for (float& v : h_Wk) v = dis(gen);
    for (float& v : h_Wv) v = dis(gen);
    for (float& v : h_Wo) v = dis(gen);
    for (float& v : h_grad_output) v = dis(gen);

    float *d_input, *d_Wq, *d_Wk, *d_Wv, *d_Wo;
    float *d_Q, *d_K, *d_V;
    float *d_scores, *d_softmax_scores, *d_attn_heads_output, *d_concat_heads, *d_output;
    float *d_softmax_max, *d_softmax_sum;
    float *d_grad_output, *d_grad_concat_heads, *d_grad_Wo, *d_grad_attn_heads_output;
    float *d_grad_V, *d_grad_softmax_scores, *d_grad_scores, *d_grad_Q, *d_grad_K;
    float *d_grad_Wq, *d_grad_Wk, *d_grad_Wv, *d_grad_input;
    float *d_grad_input_from_Q, *d_grad_input_from_K, *d_grad_input_from_V;

    CHECK_CUDA(hipMalloc(&d_input, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wq, h_Wq.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wk, h_Wk.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wv, h_Wv.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wo, h_Wo.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Q, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_K, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_V, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_scores, batch_size * num_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_softmax_scores, batch_size * num_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_attn_heads_output, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_concat_heads, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_softmax_max, batch_size * num_heads * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_softmax_sum, batch_size * num_heads * seq_len * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_grad_output, h_grad_output.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_concat_heads, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wo, h_Wo.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_attn_heads_output, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_V, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_softmax_scores, batch_size * num_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_scores, batch_size * num_heads * seq_len * seq_len * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Q, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_K, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wq, h_Wq.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wk, h_Wk.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_Wv, h_Wv.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input_from_Q, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input_from_K, h_input.size() * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_grad_input_from_V, h_input.size() * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wq, h_Wq.data(), h_Wq.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wk, h_Wk.data(), h_Wk.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wv, h_Wv.data(), h_Wv.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wo, h_Wo.data(), h_Wo.size() * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_grad_output, h_grad_output.data(), h_grad_output.size() * sizeof(float), hipMemcpyHostToDevice));
    
    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));

    const float alpha = 1.0f, beta = 0.0f;
    const int N_in = batch_size * seq_len;

    // Forward Pass
    std::cout << "\n--- 1. Forward Pass ---\n" << std::endl;
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wq, embed_dim, d_input, embed_dim, &beta, d_Q, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wk, embed_dim, d_input, embed_dim, &beta, d_K, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wv, embed_dim, d_input, embed_dim, &beta, d_V, embed_dim));
    CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, seq_len, seq_len, head_dim, &alpha, d_K, head_dim, seq_len * head_dim, d_Q, head_dim, seq_len * head_dim, &beta, d_scores, seq_len, seq_len * seq_len, batch_size * num_heads));
    const float scale_factor = 1.0f / sqrtf((float)head_dim);
    scale_kernel<<<(batch_size * num_heads * seq_len * seq_len + 255) / 256, 256>>>(d_scores, scale_factor, batch_size * num_heads * seq_len * seq_len);
    CHECK_CUDA(hipMemcpy(d_softmax_scores, d_scores, batch_size * num_heads * seq_len * seq_len * sizeof(float), hipMemcpyDeviceToDevice));
    int softmax_rows = batch_size * num_heads * seq_len;
    find_max_kernel<<<softmax_rows, 256, 256 * sizeof(float)>>>(d_softmax_scores, d_softmax_max, softmax_rows, seq_len);
    exp_sum_normalize_kernel<<<softmax_rows, 256, 256 * sizeof(float)>>>(d_softmax_scores, d_softmax_max, d_softmax_sum, softmax_rows, seq_len);
    CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, head_dim, seq_len, seq_len, &alpha, d_V, head_dim, seq_len * head_dim, d_softmax_scores, seq_len, seq_len * seq_len, &beta, d_attn_heads_output, head_dim, seq_len * head_dim, batch_size * num_heads));
    dim3 concat_blocks(batch_size * seq_len);
    dim3 concat_threads(32, 8);
    concat_heads_kernel<<<concat_blocks, concat_threads>>>(d_attn_heads_output, d_concat_heads, batch_size, num_heads, seq_len, head_dim);
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wo, embed_dim, d_concat_heads, embed_dim, &beta, d_output, embed_dim));
    
    // Backward Pass
    std::cout << "\n--- 2. Backward Pass ---\n" << std::endl;

    // Bwd Step 1: Final Projection Layer
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, embed_dim, N_in, &alpha, d_grad_output, embed_dim, d_concat_heads, embed_dim, &beta, d_grad_Wo, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, embed_dim, N_in, embed_dim, &alpha, d_Wo, embed_dim, d_grad_output, embed_dim, &beta, d_grad_concat_heads, embed_dim));
    
    // Bwd Step 2: Concatenate Heads
    split_heads_grad_kernel<<<concat_blocks, concat_threads>>>(d_grad_concat_heads, d_grad_attn_heads_output, batch_size, num_heads, seq_len, head_dim);

    // Bwd Step 3: d(Scores @ V) -> dV, dSoftmaxScores
    CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, seq_len, seq_len, head_dim, &alpha, d_grad_attn_heads_output, head_dim, seq_len * head_dim, d_V, head_dim, seq_len * head_dim, &beta, d_grad_softmax_scores, seq_len, seq_len * seq_len, batch_size * num_heads));
    CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, head_dim, seq_len, seq_len, &alpha, d_grad_attn_heads_output, head_dim, seq_len * head_dim, d_softmax_scores, seq_len, seq_len * seq_len, &beta, d_grad_V, head_dim, seq_len * head_dim, batch_size * num_heads));
    
    // Bwd Step 4: d(Softmax) -> dScores
    softmax_backward_kernel<<<softmax_rows, 256, 256 * sizeof(float)>>>(d_grad_scores, d_softmax_scores, d_grad_softmax_scores, softmax_rows, seq_len);
    
    // Bwd Step 5: d(Scaling)
    scale_kernel<<<(batch_size * num_heads * seq_len * seq_len + 255) / 256, 256>>>(d_grad_scores, scale_factor, batch_size * num_heads * seq_len * seq_len);
    
    // Bwd Step 6: d(Q @ K^T) -> dQ, dK
    CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, head_dim, seq_len, seq_len, &alpha, d_K, head_dim, seq_len * head_dim, d_grad_scores, seq_len, seq_len * seq_len, &beta, d_grad_Q, head_dim, seq_len * head_dim, batch_size * num_heads));
    CHECK_CUBLAS(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, head_dim, seq_len, seq_len, &alpha, d_Q, head_dim, seq_len * head_dim, d_grad_scores, seq_len, seq_len * seq_len, &beta, d_grad_K, head_dim, seq_len * head_dim, batch_size * num_heads));
    
    // Bwd Step 7: d(Input @ W) -> dWq, dWk, dWv, dInput
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embed_dim, embed_dim, N_in, &alpha, d_grad_Q, embed_dim, d_input, embed_dim, &beta, d_grad_Wq, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embed_dim, embed_dim, N_in, &alpha, d_grad_K, embed_dim, d_input, embed_dim, &beta, d_grad_Wk, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, embed_dim, embed_dim, N_in, &alpha, d_grad_V, embed_dim, d_input, embed_dim, &beta, d_grad_Wv, embed_dim));
    
    // Sum gradients for the input
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wq, embed_dim, d_grad_Q, embed_dim, &beta, d_grad_input_from_Q, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wk, embed_dim, d_grad_K, embed_dim, &beta, d_grad_input_from_K, embed_dim));
    CHECK_CUBLAS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, embed_dim, N_in, embed_dim, &alpha, d_Wv, embed_dim, d_grad_V, embed_dim, &beta, d_grad_input_from_V, embed_dim));
    add_matrices_kernel<<<(h_input.size() + 255) / 256, 256>>>(d_grad_input, d_grad_input_from_Q, d_grad_input_from_K, d_grad_input_from_V, h_input.size());

    std::cout << "Backward pass complete." << std::endl;
    print_matrix("Gradient w.r.t. Wo", d_grad_Wo, embed_dim, embed_dim);
    print_matrix("Gradient w.r.t. Wq", d_grad_Wq, embed_dim, embed_dim);
    print_matrix("Gradient w.r.t. Input", d_grad_input, N_in, embed_dim);

    CHECK_CUBLAS(hipblasDestroy(cublas_handle));

    return 0;
}
