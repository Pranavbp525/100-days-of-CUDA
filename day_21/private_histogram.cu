#include <iostream>
#include <hip/hip_runtime.h>
#include <string>

using namespace std;


#define NUM_BINS 7  


__global__ void private_histogram(char *data, int length, int *hist) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < length) {
        int pos = data[i] - 'a';
        if (pos >= 0 && pos < 26) {
            atomicAdd(&(hist[blockIdx.x*NUM_BINS + pos / 4]), 1);  
        }
    }

    if(blockIdx.x>0){
        __syncthreads();
        for(int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x){
            int binVal = hist[blockIdx.x*NUM_BINS + bin];
            if(binVal>0){
                atomicAdd(&(hist[bin]), binVal);
            }
        }
    }
}


void private_histogram_host(char *data_h, int length, int *hist_h) {
    int inp_size = sizeof(char) * length;
    int block_size = 256; 
    int grid_size = (length + block_size - 1) / block_size; 
    int hist_size = sizeof(int) * NUM_BINS * grid_size; 

    char *data_d;
    int *hist_d;

    hipMalloc((void**)&data_d, inp_size);
    hipMalloc((void**)&hist_d, hist_size);

    hipMemcpy(data_d, data_h, inp_size, hipMemcpyHostToDevice);
    hipMemset(hist_d, 0, hist_size); 
    private_histogram<<<grid_size, block_size>>>(data_d, length, hist_d);

    hipDeviceSynchronize();

    hipMemcpy(hist_h, hist_d, sizeof(int) * NUM_BINS, hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(hist_d);
}

int main() {
    string input_text = "one hundred days of cuda challenge";
    int length = input_text.length();

    char *data_h = new char[length];
    int *hist_h = new int[NUM_BINS](); 

    memcpy(data_h, input_text.c_str(), length);

    private_histogram_host(data_h, length, hist_h);

    cout << "Histogram of lowercase letters (grouped by 4):" << endl;
    for (int i = 0; i < NUM_BINS; i++) {
        char start_letter = 'a' + i * 4;
        char end_letter = (i == NUM_BINS - 1) ? 'z' : start_letter + 3;
        cout << "Bin " << i << " (" << start_letter << "-" << end_letter << "): " << hist_h[i] << endl;
    }

    delete[] data_h;
    delete[] hist_h;

    return 0;
}