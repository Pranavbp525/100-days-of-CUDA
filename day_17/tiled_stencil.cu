#include "hip/hip_runtime.h"
#incude <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define IN_TILE_DIM 5
#define OUT_TILE_DIM 4

__global__ void tiled_stencil(float * inp, float * out, int side){

    int i = threadIdx.z + blockIdx.z*OUT_TILE_DIM - 1;
    int j = threadIdx.y + blockIdx.y*OUT_TILE_DIM - 1;
    int k = threadIdx.x + blockIdx.x*OUT_TILE_DIM - 1;

    __shared__ float inp_s[IN_TILE_DIM][IN_TILE_DIM][IN_TILE_DIM];

    if(i>=0 && i<side && j>=0 && j<side && k>=0 && k<side){
        inp_s[threadIdx.z][threadIdx.y][threadIdx.x] = inp[i*side*side + j*side + k];
    }

    __syncthreads();

    if(i>=1 && i<side-1 && j>=1 && j<side-1 && k>=1 && k<side-1){
        if(threadIdx.x>=1 && threadIdx.x<IN_TILE_DIM-1 && threadIdx.y>=1 && threadIdx.y<IN_TILE_DIM-1 && threadIdx.z>=1 && threadIdx.z<IN_TILE_DIM-1){
            out[i*side*side + j*side + k] = c0*inp_s[threadIdx.z][threadIdx.y][threadIdx.x];
                                            + c1*inp_s[threadIdx.z][threadIdx.y][threadIdx.x-1];
                                            + c2*inp_s[threadIdx.z][threadIdx.y][threadIdx.x+1];
                                            + c3*inp_s[threadIdx.z][threadIdx.y-1][threadIdx.x];
                                            + c4*inp_s[threadIdx.z][threadIdx.y+1][threadIdx.x];
                                            + c5*inp_s[threadIdx.z-1][threadIdx.y][threadIdx.x];
                                            + c6*inp_s[threadIdx.z+1][threadIdx.y][threadIdx.x];;
                            
        }
    }

}

void tiled_stencil_host(float *inp_h, float *out_h, int side) {
    int size = side * side * side;
    int bytes = size * sizeof(float);

    
    float *inp_d, *out_d;
    hipMalloc(&inp_d, bytes);
    hipMalloc(&out_d, bytes);

    
    hipMemcpy(inp_d, inp_h, bytes, hipMemcpyHostToDevice);

    
    hipMemset(out_d, 0, bytes);

    
    dim3 blockDim(IN_TILE_DIM, IN_TILE_DIM, IN_TILE_DIM);
    dim3 gridDim((side + OUT_TILE_DIM - 1) / OUT_TILE_DIM,
                 (side + OUT_TILE_DIM - 1) / OUT_TILE_DIM,
                 (side + OUT_TILE_DIM - 1) / OUT_TILE_DIM);

    
    tiled_stencil<<<gridDim, blockDim>>>(inp_d, out_d, side);

   
    hipDeviceSynchronize();

    
    hipMemcpy(out_h, out_d, bytes, hipMemcpyDeviceToHost);

    
    hipFree(inp_d);
    hipFree(out_d);
}

int main() {
    const int side = 16; 
    int size = side * side * side;

    
    float *inp_h = new float[size];
    float *out_h = new float[size];

    
    for (int i = 0; i < size; i++) {
        inp_h[i] = static_cast<float>(i);
    }

    
    float h_c0 = 1.0f; 
    float h_c1 = 0.5f; 
    float h_c2 = 0.5f; 
    float h_c3 = 0.5f; 
    float h_c4 = 0.5f; 
    float h_c5 = 0.5f;
    float h_c6 = 0.5f;

    hipMemcpyToSymbol(HIP_SYMBOL(c0), &h_c0, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c1), &h_c1, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c2), &h_c2, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c3), &h_c3, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c4), &h_c4, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c5), &h_c5, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c6), &h_c6, sizeof(float));

    
    tiled_stencil_host(inp_h, out_h, side);

    
    cout << "Sample output (interior points: i=1, j=1, k=1 to 5):" << endl;
    for (int k = 1; k <= 5; k++) {
        int i = 1, j = 1;
        int index = i * side * side + j * side + k;
        cout << out_h[index] << " ";
    }
    cout << endl;

    
    delete[] inp_h;
    delete[] out_h;

    return 0;
}