#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define BLOCKDIM 512
#define COARSEFACTOR 2

__global__ void segmentedSumReductionKernel(float *input, float *out) {
    __shared__ float input_s[BLOCKDIM];
    unsigned int segment = COARSEFACTOR * 2 * blockDim.x * blockIdx.x;
    unsigned int i = segment + threadIdx.x;
    unsigned int t = threadIdx.x;
    float sum = input[i];
    for (unsigned int tile = 1; tile < 2 * COARSEFACTOR; ++tile) {
        sum += input[i + tile * BLOCKDIM];
    }
    input_s[t] = sum;
    for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (t < stride) {
            input_s[t] += input_s[t + stride];
        }
    }
    if (t == 0) {
        atomicAdd(out, input_s[0]);
    }
}

void segmentedSumReductionHost(float *inp_h, float *out_h, int length) {
    int block_size = BLOCKDIM; 
    int elements_per_block = 2 * COARSEFACTOR * block_size;  
    int grid_size = (length + elements_per_block - 1) / elements_per_block;

    if ((block_size & (block_size - 1)) != 0) {
        cout << "Block size must be a power of 2." << endl;
        return;
    }
    if (length % elements_per_block != 0) {
        cout << "Input length should be a multiple of " << elements_per_block << " for this implementation." << endl;
    }

    float *inp_d, *out_d;
    hipMalloc(&inp_d, length * sizeof(float));
    hipMalloc(&out_d, sizeof(float));

    hipMemcpy(inp_d, inp_h, length * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(out_d, 0, sizeof(float));

    segmentedSumReductionKernel<<<grid_size, block_size>>>(inp_d, out_d);

    hipMemcpy(out_h, out_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inp_d);
    hipFree(out_d);
}

int main() {
    const int block_size = BLOCKDIM; 
    const int grid_size = 4;
    const int length = COARSEFACTOR * 2 * block_size * grid_size;  

    float *inp_h = new float[length];
    float out_h = 0.0f;

    for (int i = 0; i < length; i++) {
        inp_h[i] = 1.0f;
    }

    segmentedSumReductionHost(inp_h, &out_h, length);

    cout << "Sum of " << length << " elements: " << out_h << endl;

    delete[] inp_h;
    return 0;
}