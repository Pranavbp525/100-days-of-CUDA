#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
using namespace std;

// Assuming co_rank and merge_sequential are defined as in the tiled merge kernel code
__device__ int co_rank_circular(int k, int *A, int m, int *B, int n, int A_S_start, int B_S_start, int tile_size) {
    int i = k < m ? k : m;
    int j = k - i;
    int i_low = 0 > (k - n) ? 0 : (k - n);
    int j_low = 0 > (k - m) ? 0 : (k - m);
    int delta;
    bool active = true;
    while (active) {
        int i_cir = (A_S_start+i)%tile_size;
        int i_m_1_cir = (A_S_start+i-1)%tile_size;
        int j_cir = (B_S_start+j)%tile_size;
        int j_m_1_cir = (B_S_start+i-1)%tile_size;
        if (i > 0 && j < n && A[i_m_1_cir] > B[j_cir]) {
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            j = j + delta;
            i = i - delta;
        } else if (j > 0 && i < m && B[j_m_1_cir] >= A[i_cir]) {
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            i = i + delta;
            j = j - delta;
        } else {
            active = false;
        }
    }
    return i;
}

__device__ void merge_sequential_circular(int *A, int m, int *B, int n, int *C, int A_S_start, int B_S_start, int tile_size) {
    int i = 0;
    int j = 0;
    int k = 0;
    while ((i < m) && (j < n)) {
        int i_cir =(A_S_start+i)%tile_size;
        int j_cir = (B_S_start+j)%tile_size;
        if (A[i_cir] < B[j_cir]) {
            C[k++] = A[i_cir];i++;
        } else {
            C[k++] = B[j_cir];j++;
        }
    }
    if (i == m) {
        for(;j<n;j++){
            int j_cir = (B_S_start+j)%tile_size;
            C[k++] = B[j_cir];
        }
    } else {
        for(;i<m;i++){
            int i_cir = (A_S_start+i)%tile_size;
            C[k++] = A[i_cir];
        }
    }
}

__global__ void circular_tiled_merge_kernel(int *A, int m, int *B, int n, int *C, int tile_size) {
    extern __shared__ int shareAB[];
    int *A_S = &shareAB[0];
    int *B_S = &shareAB[tile_size];

    int C_curr = blockIdx.x * ceil((float)(m + n) / gridDim.x);
    int C_next = min((blockIdx.x + 1) * ceil((float)(m + n) / gridDim.x), m + n);

    if (threadIdx.x == 0) {
        A_S[0] = co_rank(C_curr, A, m, B, n);
        A_S[1] = co_rank(C_next, A, m, B, n);
    }
    __syncthreads();

    int A_curr = A_S[0];
    int A_next = A_S[1];
    int B_curr = C_curr - A_curr;
    int B_next = C_next - A_next;
    __syncthreads();

    int counter = 0;
    int C_length = C_next - C_curr;
    int A_length = A_next - A_curr;
    int B_length = B_next - B_curr;
    int total_iteration = ceil((float)C_length / tile_size);
    int C_completed = 0;
    int A_consumed = 0;
    int B_consumed = 0;
    int A_S_start = 0;
    int B_S_start = 0;
    int A_S_consumed = tile_size;
    int B_S_consumed = tile_size;
    
    while (counter < total_iteration) {
        for (int i = 0; i < A_S_consumed; i += blockDim.x) {
            if (i + threadIdx.x < A_length - A_consumed && (i + threadIdx.x)<A_S_consumed) {
                A_S[(A_S_start +(tile_size - A_S_consumed)+i+threadIdx.x)%tile_size] = A[A_curr + A_consumed + i + threadIdx.x];
            }
        }
        for (int i = 0; i < B_S_consumed; i += blockDim.x) {
            if (i + threadIdx.x < B_length - B_consumed && (i + threadIdx.x)<B_S_consumed) {
                B_S[(B_S_start +(tile_size - B_S_consumed)+i+threadIdx.x)%tile_size] = A[A_curr + A_consumed + i + threadIdx.x];
            }
        }
        __syncthreads();

        int c_curr = threadIdx.x * (tile_size / blockDim.x);
        int c_next = (threadIdx.x + 1) * (tile_size / blockDim.x);
        c_curr = (c_curr <= C_length - C_completed) ? c_curr : C_length - C_completed;
        c_next = (c_next <= C_length - C_completed) ? c_next : C_length - C_completed;
        int a_curr = co_rank_circular(c_curr, A_S, min(tile_size, A_length - A_consumed), B_S, min(tile_size, B_length - B_consumed), A_S_start,B_S_start, tile_size);
        int b_curr = c_curr - a_curr;
        int a_next = co_rank_circular(c_next, A_S, min(tile_size, A_length - A_consumed), B_S, min(tile_size, B_length - B_consumed), A_S_start,B_S_start, tile_size);
        int b_next = c_next - a_next;

        merge_sequential_circular(A_S, a_next - a_curr, B_S, b_next - b_curr, C + C_curr + C_completed + c_curr, A_S_start+a_curr, B_S_start+b_curr, tile_size);
        counter++;
        A_S_consumed = co_rank_circular(min(tile_size, C_length-C_completed), A_S, min(tile_size, A_length-A_consumed), B_S, min(tile_size, B_lenggth-B_consumed), A_S_start, B_S_start, tile_size)
        B_S_consumed = min(tile_size, C_length-C_completed)-A_S_consumed;
        A_consumed +=A_S_consumed;

        C_completed += min(tile_size, C_length-C_completed);
        
        B_consumed = C_completed - A_consumed;
        A_S_start = (B_S_start + B_S_consumed)%tile_size;
        B_S_start = (A_S_start + B_S_consumed)%tile_size;
        __syncthreads();
    }
}

void parallel_merge_host(int *A_h, int m, int *B_h, int n, int *C_h) {
    int *A_d, *B_d, *C_d;
    size_t size_A = m * sizeof(int);
    size_t size_B = n * sizeof(int);
    size_t size_C = (m + n) * sizeof(int);

    hipMalloc(&A_d, size_A);
    hipMalloc(&B_d, size_B);
    hipMalloc(&C_d, size_C);

    hipMemcpy(A_d, A_h, size_A, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size_B, hipMemcpyHostToDevice);

    const int block_size = 256;
    const int tile_size = 1024; // Adjustable parameter based on device capabilities
    int grid_size = (m + n + tile_size - 1) / tile_size;
    if (grid_size == 0) grid_size = 1; // Ensure at least one block

    size_t shared_mem_size = 2 * tile_size * sizeof(int);

    tiled_merge_kernel<<<grid_size, block_size, shared_mem_size>>>(A_d, m, B_d, n, C_d, tile_size);

    hipMemcpy(C_h, C_d, size_C, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    const int m = 5;
    const int n = 5;
    int A_h[m] = {1, 3, 5, 7, 9};
    int B_h[n] = {2, 4, 6, 8, 10};
    int C_h[m + n];

    parallel_merge_host(A_h, m, B_h, n, C_h);

    cout << "Merged Array: ";
    for (int i = 0; i < m + n; i++) {
        cout << C_h[i] << " ";
    }
    cout << endl;

    return 0;
}