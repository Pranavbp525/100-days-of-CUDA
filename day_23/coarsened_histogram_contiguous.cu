#include <iostream>
#include <hip/hip_runtime.h>
#include <string>

using namespace std;

#define NUM_BINS 7  
#define CFACTOR 3   
__global__ void private_histogram(char *data, int length, int *hist) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ int hist_s[NUM_BINS];

    for (int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        hist_s[bin] = 0;
    }
    __syncthreads();

    for (int i = tid * CFACTOR; i < min((tid + 1) * CFACTOR, length); i++) {
        int pos = data[i] - 'a';
        if (pos >= 0 && pos < 26) {  
            atomicAdd(&(hist_s[pos / 4]), 1); 
        }
    }

    __syncthreads();

    for (int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
        int binVal = hist_s[bin];
        if (binVal > 0) {
            atomicAdd(&(hist[bin]), binVal);
        }
    }
}

void private_histogram_host(char *data_h, int length, int *hist_h) {
    int inp_size = sizeof(char) * length;
    int hist_size = sizeof(int) * NUM_BINS;

    char *data_d;
    int *hist_d;

    hipMalloc((void**)&data_d, inp_size);
    hipMalloc((void**)&hist_d, hist_size);

    hipMemcpy(data_d, data_h, inp_size, hipMemcpyHostToDevice);
    hipMemset(hist_d, 0, hist_size);

    int block_size = 256;  
    int total_threads = (length + CFACTOR - 1) / CFACTOR;  
    int grid_size = (total_threads + block_size - 1) / block_size;  

    private_histogram<<<grid_size, block_size>>>(data_d, length, hist_d);

    hipDeviceSynchronize();

    hipMemcpy(hist_h, hist_d, hist_size, hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(hist_d);
}

int main() {
    string input_text = "one hundred days of cuda challenge";
    int length = input_text.length();

    char *data_h = new char[length];
    int *hist_h = new int[NUM_BINS]();  

    memcpy(data_h, input_text.c_str(), length);

    private_histogram_host(data_h, length, hist_h);

    cout << "Histogram of lowercase letters (grouped by 4):" << endl;
    for (int i = 0; i < NUM_BINS; i++) {
        char start_letter = 'a' + i * 4;
        char end_letter = (i == NUM_BINS - 1) ? 'z' : start_letter + 3;
        cout << "Bin " << i << " (" << start_letter << "-" << end_letter << "): " << hist_h[i] << endl;
    }

    delete[] data_h;
    delete[] hist_h;

    return 0;
}